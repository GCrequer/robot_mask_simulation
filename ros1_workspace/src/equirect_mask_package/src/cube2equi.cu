
#include <hip/hip_runtime.h>
extern "C" __global__
void cube2equi(const uchar3* posy, const uchar3* negx, const uchar3* posx,
               const uchar3* negz, const uchar3* negy, const uchar3* posz,
               uchar3* dst, int rows, int cols, int dims)
{
    float PI = 3.14159265358979323846;

    int dst_x = blockDim.x * blockIdx.x + threadIdx.x;
    int dst_y = blockDim.y * blockIdx.y + threadIdx.y;

    if (dst_x < cols && dst_y < rows)
    {
        float v = 1.0 - ((float(dst_y)) / (dims * 2));
        float phi = v * PI;

        float u = (float(dst_x)) / (dims * 4);
        float theta = u * 2 * PI;

        float x = cos(theta) * sin(phi);
        float y = sin(theta) * sin(phi);
        float z = cos(phi);

        float a = fmaxf(fmaxf(fabsf(x), fabsf(y)), fabsf(z));

        float xx = x / a;
        float yy = y / a;
        float zz = z / a;

        float xPixel, yPixel, yTemp, imageSelect;

        if (yy == -1)
        {
            xPixel = (((-1.0 * tan(atan(x / y)) + 1.0) / 2.0) * dims);
            yTemp = (((-1.0 * tan(atan(z / y)) + 1.0) / 2.0) * (dims - 1.0));
            imageSelect = 1;
        }
        else if (xx == 1)
        {
            xPixel = (((tan(atan(y / x)) + 1.0) / 2.0) * dims);
            yTemp = (((tan(atan(z / x)) + 1.0) / 2.0) * dims);
            imageSelect = 2;
        }
        else if (yy == 1)
        {
            xPixel = (((-1 * tan(atan(x / y)) + 1.0) / 2.0) * dims);
            yTemp = (((tan(atan(z / y)) + 1.0) / 2.0) * (dims - 1));
            imageSelect = 3;
        }
        else if (xx == -1) {
            xPixel = (((tan(atan(y / x)) + 1.0) / 2.0) * dims);
            yTemp = (((-1 * tan(atan(z / x)) + 1.0) / 2.0) * (dims - 1));
            imageSelect = 4;
        }
        else if (zz == 1)
        {
            xPixel = (((tan(atan(y / z)) + 1.0) / 2.0) * dims);
            yTemp = (((-1 * tan(atan(x / z)) + 1.0) / 2.0) * (dims - 1));
            imageSelect = 5;
        }
        else if (zz == -1)
        {
            xPixel = (((-1 * tan(atan(y / z)) + 1.0) / 2.0) * dims);
            yTemp = (((-1 * tan(atan(x / z)) + 1.0) / 2.0) * (dims - 1));
            imageSelect = 6;
        }

        yPixel = yTemp > dims - 1 ? (dims - 1) : yTemp;

        if (yPixel > dims - 1)
            yPixel = dims - 1;
        if (xPixel > dims - 1)
            xPixel = dims - 1;

        uchar3 value;
        if (imageSelect == 1)
            value = posy[int(yPixel) * dims + int(xPixel)];
        else if (imageSelect == 2)
            value = posx[int(yPixel) * dims + int(xPixel)];
        else if (imageSelect == 3)
            value = negy[int(yPixel) * dims + int(xPixel)];
        else if (imageSelect == 4)
            value = negx[int(yPixel) * dims + int(xPixel)];
        else if (imageSelect == 5)
            value = negz[int(yPixel) * dims + int(xPixel)];
        else if (imageSelect == 6)
            value = posz[int(yPixel) * dims + int(xPixel)];

        dst[dst_y * cols + dst_x] = value;
    }
}